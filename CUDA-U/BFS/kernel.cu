#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016 University of Cordoba and University of Illinois
 * All rights reserved.
 *
 * Developed by:    IMPACT Research Group
 *                  University of Cordoba and University of Illinois
 *                  http://impact.crhc.illinois.edu/
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * with the Software without restriction, including without limitation the 
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *      > Redistributions of source code must retain the above copyright notice,
 *        this list of conditions and the following disclaimers.
 *      > Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimers in the
 *        documentation and/or other materials provided with the distribution.
 *      > Neither the names of IMPACT Research Group, University of Cordoba, 
 *        University of Illinois nor the names of its contributors may be used 
 *        to endorse or promote products derived from this Software without 
 *        specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
 * THE SOFTWARE.
 *
 */

#define _CUDA_COMPILER_

#include "support/common.h"
#include <cstdio>

// CUDA kernel ------------------------------------------------------------------------------------------
__global__ void BFS_gpu(Node *graph_nodes_av, Edge *graph_edges_av, int *cost,
    int *color, int *q1, int *q2, int *n_t,
    int *head, int *tail, int *threads_end,
    int *threads_run, int *overflow, int LIMIT, const int CPU) {

    extern __shared__ int l_mem[];
    int* base = l_mem;
    
    const int tid     = threadIdx.x;
    const int gtid    = blockIdx.x * blockDim.x + threadIdx.x;
    const int MAXWG   = gridDim.x;
    const int WG_SIZE = blockDim.x;

    int *qin, *qout;

    int iter = 1;

    //if (gtid == 0)
    //    printf("Iter 0, n_t %d, CPU %d\n", *n_t, CPU);

    while(*n_t != 0) {

        // Swap queues
        if(iter % 2 == 0) {
            qin  = q1;
            qout = q2;
        } else {
            qin  = q2;
            qout = q1;
        }

        if((*n_t >= LIMIT) | (CPU == 0)) {

            // Fetch frontier elements from the queue
            if(tid == 0)
                *base = atomicAdd_system(&head[0], WG_SIZE);
            __syncthreads();

            int my_base = *base;
            while(my_base < *n_t) {
                if(my_base + tid < *n_t && *overflow == 0) {
                    // Visit a node from the current frontier
                    int pid = qin[my_base + tid];
                    //////////////// Visit node ///////////////////////////
                    atomicExch_system(&cost[pid], iter); // Node visited

                    Node cur_node;
                    cur_node.x = graph_nodes_av[pid].x;
                    cur_node.y = graph_nodes_av[pid].y;
                    // For each outgoing edge
                    for(int i = cur_node.x; i < cur_node.y + cur_node.x; i++) {
                        int id        = graph_edges_av[i].x;
                        int old_color = atomicMax_system(&color[id], BLACK);
                        if(old_color < BLACK) {
                            // Push to the queue
                            int index_o       = atomicAdd_system(&tail[0],1);
                            qout[index_o] = id;
                        }
                    }
                }
                if(tid == 0)
                    *base = atomicAdd_system(&head[0], WG_SIZE); // Fetch more frontier elements from the queue
                __syncthreads();
                my_base = *base;
            }
        }

        if(CPU) { // if CPU is available
            iter++;
            if(tid == 0) {
                atomicAdd_system(&threads_end[0], WG_SIZE);

                __threadfence();

                while(atomicAdd_system(&threads_run[0], 0) < iter) {
                }
            }
        } else { // if GPU only
            iter++;
            if(tid == 0){
                atomicAdd_system(&threads_end[0], WG_SIZE);
            }
            __threadfence();
            if(gtid == 0) {
                while(atomicAdd_system(&threads_end[0], 0) != MAXWG * WG_SIZE) {
                }
                *n_t = atomicAdd_system(&tail[0], 0);
                atomicExch_system(&tail[0], 0);
                atomicExch_system(&head[0], 0);
                atomicExch_system(&threads_end[0], 0);
                atomicAdd_system(&threads_run[0], 1);
                //printf("Iter %d, n_t %d\n",iter, *n_t);
            }
            if(tid == 0 && gtid != 0) {
                while(atomicAdd_system(&threads_run[0], 0) < iter) {
                }
            }
        }
        __syncthreads();
    }
}

hipError_t call_BFS_gpu(int blocks, int threads, Node *graph_nodes_av, Edge *graph_edges_av, int *cost,
    int *color, int *q1, int *q2, int *n_t,
    int *head, int *tail, int *threads_end, int *threads_run,
    int *overflow, int LIMIT, const int CPU, int l_mem_size){

    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    BFS_gpu<<<dimGrid, dimBlock, l_mem_size>>>(graph_nodes_av, graph_edges_av, cost,
        color, q1, q2, n_t,
        head, tail, threads_end, threads_run,
        overflow, LIMIT, CPU);
    
    hipError_t err = hipGetLastError();
    return err;
}
