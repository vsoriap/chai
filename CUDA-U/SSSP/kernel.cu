#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016 University of Cordoba and University of Illinois
 * All rights reserved.
 *
 * Developed by:    IMPACT Research Group
 *                  University of Cordoba and University of Illinois
 *                  http://impact.crhc.illinois.edu/
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * with the Software without restriction, including without limitation the 
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *      > Redistributions of source code must retain the above copyright notice,
 *        this list of conditions and the following disclaimers.
 *      > Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimers in the
 *        documentation and/or other materials provided with the distribution.
 *      > Neither the names of IMPACT Research Group, University of Cordoba, 
 *        University of Illinois nor the names of its contributors may be used 
 *        to endorse or promote products derived from this Software without 
 *        specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
 * THE SOFTWARE.
 *
 */

#define _CUDA_COMPILER_

#include "support/common.h"
#include <cstdio>

// CUDA kernel ------------------------------------------------------------------------------------------
__global__ void SSSP_gpu(Node *graph_nodes_av, Edge *graph_edges_av, int *cost,
    int *color, int *q1, int *q2, int *n_t,
    int *head, int *tail, int *threads_end,
    int *threads_run, int *overflow, int *gray_shade,
    int LIMIT, int CPU) {

    extern __shared__ int l_mem[];
    int* base = l_mem;

    const int tid     = threadIdx.x;
    const int gtid    = blockIdx.x * blockDim.x + threadIdx.x;
    const int MAXWG   = gridDim.x;
    const int WG_SIZE = blockDim.x;

    int *qin, *qout;

    int iter = 1;

    while(*n_t != 0) {

        // Swap queues
        if(iter % 2 == 0) {
            qin  = q1;
            qout = q2;
        } else {
            qin  = q2;
            qout = q1;
        }

        if((*n_t >= LIMIT) | (CPU == 0)) {

            int gray_shade_local = atomicAdd_system(&gray_shade[0], 0);

            // Fetch frontier elements from the queue
            if(tid == 0)
                *base = atomicAdd_system(&head[0], WG_SIZE);
            __syncthreads();

            int my_base = *base;
            while(my_base < *n_t) {
                if(my_base + tid < *n_t && *overflow == 0) {
                    // Visit a node from the current frontier
                    int pid = qin[my_base + tid];
                    //////////////// Visit node ///////////////////////////
                    atomicExch_system(&color[pid], BLACK); // Node visited
                    int  cur_cost = atomicAdd_system(&cost[pid], 0); // Look up shortest-path distance to this node
                    Node cur_node;
                    cur_node.x = graph_nodes_av[pid].x;
                    cur_node.y = graph_nodes_av[pid].y;
                    Edge cur_edge;
                    // For each outgoing edge
                    for(int i = cur_node.x; i < cur_node.y + cur_node.x; i++) {
                        cur_edge.x = graph_edges_av[i].x;
                        cur_edge.y = graph_edges_av[i].y;
                        int id     = cur_edge.x;
                        int cost_local   = cur_edge.y;
                        cost_local += cur_cost;
                        int orig_cost = atomicMax_system(&cost[id], cost_local);
                        if(orig_cost < cost_local) {
                            int old_color = atomicMax_system(&color[id], gray_shade_local);
                            if(old_color != gray_shade_local) {
                                // Push to the queue
                                int index_o   = atomicAdd_system(&tail[0],1);
                                qout[index_o] = id;
                            }
                        }
                    }
                }
                if(tid == 0)
                    *base = atomicAdd_system(&head[0], WG_SIZE); // Fetch more frontier elements from the queue
                __syncthreads();
                my_base = *base;
            }
        }

        if(CPU) { // if CPU is available
            iter++;
            if(tid == 0) {
                atomicAdd_system(&threads_end[0], WG_SIZE);

                __threadfence();

                while(atomicAdd_system(&threads_run[0], 0) < iter) {
                }
            }
        } else { // if GPU only
            iter++;
            if(tid == 0)
                atomicAdd_system(&threads_end[0], WG_SIZE);
            __threadfence();
            if(gtid == 0) {
                while(atomicAdd_system(&threads_end[0], 0) != MAXWG * WG_SIZE) {
                }
                *n_t = atomicAdd_system(&tail[0], 0);
                atomicExch_system(&tail[0], 0);
                atomicExch_system(&head[0], 0);
                atomicExch_system(&threads_end[0], 0);
                if(iter % 2 == 0)
                    atomicExch_system(&gray_shade[0], GRAY0);
                else
                    atomicExch_system(&gray_shade[0], GRAY1);
                __threadfence();
                atomicAdd_system(&threads_run[0], 1);
                //printf("Iter %d, n_t %d\n",iter, *n_t);
            }
            if(tid == 0 && gtid != 0) {
                while(atomicAdd_system(&threads_run[0], 0) < iter) {
                }
            }
        }
        __syncthreads();
    }
}

hipError_t call_SSSP_gpu(int blocks, int threads, Node *graph_nodes_av, Edge *graph_edges_av, int *cost,
    int *color, int *q1, int *q2, int *n_t,
    int *head, int *tail, int *threads_end, int *threads_run,
    int *overflow, int *gray_shade, int LIMIT, const int CPU, int l_mem_size){

    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    SSSP_gpu<<<dimGrid, dimBlock, l_mem_size>>>(graph_nodes_av, graph_edges_av, cost,
        color, q1, q2, n_t,
        head, tail, threads_end, threads_run,
        overflow, gray_shade, LIMIT, CPU);
    
    hipError_t err = hipGetLastError();
    return err;
}
